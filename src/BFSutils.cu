#include <hip/hip_runtime.h>

#include <iostream>
#define NUM_THREADS 16
// ========================= Parallel BFS ============================= //

__global__ void BFS_kernel(int N, int level, int *devV, int *devE, int *devD, int *devP, int *devFlag) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    int valueChange = 0;
    if (thid < N && devD[thid] == level) {
        int u = thid;
        for (int i = 1; i <= devE[devV[u]]; i++) {
            int v = devE[devV[u]+i];
            if (level + 1 < devD[v]) {
                devD[v] = level + 1;
                devP[v] = i;
                valueChange = 1;
            }
        }
    }
    if (valueChange) {
        *devFlag = 1;
    }
}

// ========================= Queue BFS ============================ //

__global__ void queueBfs(int level, int *devV, int *devE, int *devD, int *devP,
              int queueSize, int *nextQueueSize, int *devCurrentQueue, int *devNextQueue) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;

    if (thid < queueSize) {
        int u = devCurrentQueue[thid];
        for (int i = 1; i <= devE[devV[u]]; i++) {
            int v = devE[devV[u]+i];
            if (devD[v] == INT_MAX && atomicMin(&devD[v], level + 1) == INT_MAX) {
                devP[v] = devV[u]+i;
                int position = atomicAdd(nextQueueSize, 1);
                devNextQueue[position] = v;
            }
        }
    }
}


// ========================= Scan BFS ============================= //

__global__ void nextLayer(int level, int *devV, int *devE, int *devP, int *devD, int queueSize, int *devCurrentQueue) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    if (thid < queueSize) {
        int u = devCurrentQueue[thid];
        for (int i = 1; i <= devE[devV[u]]; i++) {
            int pos = devV[u]+i;
            int v = devE[pos];
            if (level + 1 < devD[v]) {
                devD[v] = level + 1;
                devP[v] = pos;
            }
        }
    }

}

__global__ void countDegrees(int *devV, int *devE, int *devP, int queueSize, int *devCurrentQueue, int *devDegrees) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    if (thid < queueSize) {
        int u = devCurrentQueue[thid];
        int degree = 0;
        for (int i = 1; i <= devE[devV[u]]; i++) {
            int pos = devV[u]+i;
            int v = devE[pos];
            if (devP[v] == (pos) && v != u) {
                ++degree;
            }
        }
        devDegrees[thid] = degree;
    }
}

__global__ void scanDegrees(int N, int *devDegrees, int *incrDegrees, int *num) {
    //TODO: copied this part, need to understand
    int thid = blockIdx.x * blockDim.x + threadIdx.x; 
    printf("scanDegrees thid %d\n", thid);
    printf("scanDegrees thid %d\n", thid);
    // unsigned int fakeN = N;
    int val = atomicAdd(num, 1);
    printf("\n [ fakeN %d val %d ] \n", *num, val);
    if (thid < NUM_THREADS) {
        //write initial values to shared memory
        __shared__ int prefixSum[1024];
        int modulo = threadIdx.x;
        prefixSum[modulo] = devDegrees[thid];
        __syncthreads();

        //calculate scan on this block
        //go up
        for (int nodeSize = 2; nodeSize <= 1024; nodeSize <<= 1) {
            if ((modulo & (nodeSize - 1)) == 0) {
                if (thid + (nodeSize >> 1) < N) {
                    int nextPosition = modulo + (nodeSize >> 1);
                    prefixSum[modulo] += prefixSum[nextPosition];
                }
            }
            __syncthreads();
        }

        //write information for increment prefix sums
        if (modulo == 0) {
            int block = thid >> 10;
            incrDegrees[block + 1] = prefixSum[modulo];
        }

        //go down
        for (int nodeSize = 1024; nodeSize > 1; nodeSize >>= 1) {
            if ((modulo & (nodeSize - 1)) == 0) {
                if (thid + (nodeSize >> 1) < N) {
                    int next_position = modulo + (nodeSize >> 1);
                    int tmp = prefixSum[modulo];
                    prefixSum[modulo] -= prefixSum[next_position];
                    prefixSum[next_position] = tmp;

                }
            }
            __syncthreads();
        }
        devDegrees[thid] = prefixSum[modulo];
    }
}

__global__ void assignVerticesNextQueue(int *devV, int *devE, int *devP, int queueSize,
                             int *devCurrentQueue, int *devNextQueue, int *devDegrees, int *incrDegrees,
                             int nextQueueSize) {
    //TODO: copied this part, need to understand
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("assignVerticesNextQ thid %d\n", thid);
    if (thid < queueSize) {
        __shared__ int sharedIncrement;
        if (!threadIdx.x) {
            sharedIncrement = incrDegrees[thid >> 10];
        }
        __syncthreads();

        int sum = 0;
        if (threadIdx.x) {
            sum = devDegrees[thid - 1];
        }

        int u = devCurrentQueue[thid];
        int counter = 0;
        for (int i = 1; i <= devE[devV[u]]; i++) {
            int v = devE[devV[u]+i];
            if (devP[v] == devV[u]+i && v != u) {
                int nextQueuePlace = sharedIncrement + sum + counter;
                // printf("nextQplace %d u %d v %d counter %d\n", nextQueuePlace, u, v, counter);
                devNextQueue[nextQueuePlace] = v;
                counter++;
            }
        }
    }
}