#include <hip/hip_runtime.h>


#define coarsen_factor 2
#define stride 32

/*

    Thread Level Coarsening 
    With Coarsening factor of 2 
    And Stride Length of 32
    
    Corasening Factor will reamin fixed in this file 
    Stride Length macro above can be changed for testing

*/


__global__ void SSSP_kernel1(int *V, int *E, int *W, bool *M, int *C, int *U, int n) {
    
    int tid0 = ((threadIdx.x/stride)*stride)*coarsen_factor  + threadIdx.x%stride + (blockIdx.x * coarsen_factor) * blockDim.x;
    int tid1 = tid0 + stride;
    if (tid0 < n && M[tid0]) {
        M[tid0] = false;
        int pos = V[tid0], size = E[pos];
        for (int i = pos + 1; i < pos + size + 1; i++) {
            int nid = E[i];
            atomicMin(&U[nid], C[tid0] + W[i]);
        }
    }

    if (tid1 < n && M[tid1]) {
        M[tid1] = false;
        int pos = V[tid1], size = E[pos];
        for (int i = pos + 1; i < pos + size + 1; i++) {
            int nid = E[i];
            atomicMin(&U[nid], C[tid1] + W[i]);
        }
    }

}

__global__ void SSSP_kernel2(bool *M, int *C, int *U, bool *flag, int n) {
    int tid0 = ((threadIdx.x/stride)*stride)*coarsen_factor + threadIdx.x%stride + (blockIdx.x * coarsen_factor) * blockDim.x;
    int tid1 = tid0 + stride;

    if (tid0 < n) {
        if (C[tid0] > U[tid0]) {
            C[tid0] = U[tid0];
            M[tid0] = true;
            *flag = true;
        }
        U[tid0] = C[tid0];
    }

    if (tid1 < n) {
        if (C[tid1] > U[tid1]) {
            C[tid1] = U[tid1];
            M[tid1] = true;
            *flag = true;
        }
        U[tid1] = C[tid1];
    }
}