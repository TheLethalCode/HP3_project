#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../include/BFWkernels.h"
#include "../include/graph.h"
#include <iostream>

#define HANDLE_ERROR(error) { \
    if (error != hipSuccess) { \
        fprintf(stderr, "not cuda success\n"); \
        fprintf(stderr, "%s in %s at line %d\n", \
                hipGetErrorString(error), __FILE__, __LINE__); \
        exit(EXIT_FAILURE); \
    } \
} \



static __global__
void _naive_fw_kernel(const int u, size_t pitch, const int nvertex, int* const graph)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (y < nvertex && x < nvertex) 
    {
        int indexYX = y * pitch + x;
        int indexUX = u * pitch + x;

        int newPath = graph[y * pitch + u] + graph[indexUX];
        int oldPath = graph[indexYX];
        if (oldPath > newPath) 
        {
            graph[indexYX] = newPath;
        }
    }
}


static __global__
void _blocked_fw_dependent_ph(const int blockId, size_t pitch, const int nvertex, int* const graph) 
{
    __shared__ int cacheGraph[BLOCK_SIZE][BLOCK_SIZE];

    const int idx = threadIdx.x;
    const int idy = threadIdx.y;

    const int v1 = BLOCK_SIZE * blockId + idy;
    const int v2 = BLOCK_SIZE * blockId + idx;

    int newPath;

    const int cellId = v1 * pitch + v2;
    if (v1 < nvertex && v2 < nvertex) 
    {
        cacheGraph[idy][idx] = graph[cellId];
    } 
    else 
    {
        cacheGraph[idy][idx] = INF;
    }

    // Synchronize to make sure the all value are loaded in block
    __syncthreads();

    #pragma unroll
    for (int u = 0; u < BLOCK_SIZE; ++u) 
    {
        newPath = cacheGraph[idy][u] + cacheGraph[u][idx];

        // Synchronize before calculate new value
        __syncthreads();
        if (newPath < cacheGraph[idy][idx]) 
        {
            cacheGraph[idy][idx] = newPath;
        }

        // Synchronize to make sure that all value are current
        __syncthreads();
    }

    if (v1 < nvertex && v2 < nvertex) {
        graph[cellId] = cacheGraph[idy][idx];
    }
}


static __global__
void _blocked_fw_partial_dependent_ph(const int blockId, size_t pitch, const int nvertex, int* const graph)
{
    if (blockIdx.x == blockId) return;

    const int idx = threadIdx.x;
    const int idy = threadIdx.y;

    int v1 = BLOCK_SIZE * blockId + idy;
    int v2 = BLOCK_SIZE * blockId + idx;

    __shared__ int cacheGraphBase[BLOCK_SIZE][BLOCK_SIZE];

    // Load base block for graph and predecessors
    int cellId = v1 * pitch + v2;

    if (v1 < nvertex && v2 < nvertex) 
    {
        cacheGraphBase[idy][idx] = graph[cellId];
    } 
    else 
    {
        cacheGraphBase[idy][idx] = INF;
    }

    // Load i-aligned singly dependent blocks
    if (blockIdx.y == 0) 
    {
        v2 = BLOCK_SIZE * blockIdx.x + idx;
    } 
    else 
    {
   // Load j-aligned singly dependent blocks
        v1 = BLOCK_SIZE * blockIdx.x + idy;
    }

    __shared__ int cacheGraph[BLOCK_SIZE][BLOCK_SIZE];

    // Load current block for graph and predecessors
    int currentPath;

    cellId = v1 * pitch + v2;
    if (v1 < nvertex && v2 < nvertex) 
    {
        currentPath = graph[cellId];
    } 
    else 
    {
        currentPath = INF;
    }
    cacheGraph[idy][idx] = currentPath;

    // Synchronize to make sure the all value are saved in cache
    __syncthreads();

    int newPath;
    // Compute i-aligned singly dependent blocks
    if (blockIdx.y == 0) 
    {
        #pragma unroll
        for (int u = 0; u < BLOCK_SIZE; ++u) {
            newPath = cacheGraphBase[idy][u] + cacheGraph[u][idx];

            if (newPath < currentPath) 
            {
                currentPath = newPath;
            }
            // Synchronize to make sure that all threads compare new value with old
            __syncthreads();

           // Update new values
            cacheGraph[idy][idx] = currentPath;

           // Synchronize to make sure that all threads update cache
            __syncthreads();
        }
    } 
    else 
    {
    // Compute j-aligned singly dependent blocks
        #pragma unroll
        for (int u = 0; u < BLOCK_SIZE; ++u) 
        {
            newPath = cacheGraph[idy][u] + cacheGraphBase[u][idx];

            if (newPath < currentPath) 
            {
                currentPath = newPath;
            }

            // Synchronize to make sure that all threads compare new value with old
            __syncthreads();

           // Update new values
            cacheGraph[idy][idx] = currentPath;

           // Synchronize to make sure that all threads update cache
            __syncthreads();
        }
    }

    if (v1 < nvertex && v2 < nvertex) 
    {
        graph[cellId] = currentPath;
    }
}


static __global__
void _blocked_fw_independent_ph(const int blockId, size_t pitch, const int nvertex, int* const graph)//, int* const pred) 
{
    if (blockIdx.x == blockId || blockIdx.y == blockId) return;

    const int idx = threadIdx.x;
    const int idy = threadIdx.y;

    const int v1 = blockDim.y * blockIdx.y + idy;
    const int v2 = blockDim.x * blockIdx.x + idx;

    __shared__ int cacheGraphBaseRow[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int cacheGraphBaseCol[BLOCK_SIZE][BLOCK_SIZE];

    int v1Row = BLOCK_SIZE * blockId + idy;
    int v2Col = BLOCK_SIZE * blockId + idx;

    // Load data for block
    int cellId;
    if (v1Row < nvertex && v2 < nvertex) 
    {
        cellId = v1Row * pitch + v2;

        cacheGraphBaseRow[idy][idx] = graph[cellId];
    }
    else 
    {
        cacheGraphBaseRow[idy][idx] = INF;
    }

    if (v1  < nvertex && v2Col < nvertex) 
    {
        cellId = v1 * pitch + v2Col;
        cacheGraphBaseCol[idy][idx] = graph[cellId];
    }
    else 
    {
        cacheGraphBaseCol[idy][idx] = INF;
    }

    // Synchronize to make sure the all value are loaded in virtual block
   __syncthreads();

   int currentPath;
   int newPath;

   // Compute data for block
   if (v1  < nvertex && v2 < nvertex) 
   {
       cellId = v1 * pitch + v2;
       currentPath = graph[cellId];

        #pragma unroll
       for (int u = 0; u < BLOCK_SIZE; ++u) 
       {
           newPath = cacheGraphBaseCol[idy][u] + cacheGraphBaseRow[u][idx];
           if (currentPath > newPath) 
           {
               currentPath = newPath;
           }
       }
       graph[cellId] = currentPath;
   }
}


void cudaNaiveFW(int nvertex, int *graph)
{
    // Choose which GPU to run on, change this on a multi-GPU system.
    HANDLE_ERROR(hipSetDevice(0));

    // Initialize the grid and block dimensions here
    dim3 dimGrid((nvertex - 1) / BLOCK_SIZE + 1, (nvertex - 1) / BLOCK_SIZE + 1, 1);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

    int *graphDevice;
    size_t height = nvertex;
    size_t width = height*sizeof(int);
    size_t pitch;

    HANDLE_ERROR(hipMallocPitch(&graphDevice, &pitch, width, height));

    HANDLE_ERROR(hipMemcpy2D(graphDevice, pitch, graph, width, width, height, hipMemcpyHostToDevice));

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(_naive_fw_kernel), hipFuncCachePreferL1);
    for(int vertex = 0; vertex < nvertex; ++vertex) {
        _naive_fw_kernel<<<dimGrid, dimBlock>>>(vertex, pitch / sizeof(int), nvertex, graphDevice);
    }

    // Check for any errors launching the kernel
    HANDLE_ERROR(hipGetLastError());
    HANDLE_ERROR(hipDeviceSynchronize());

    HANDLE_ERROR(hipMemcpy2D(graph, width, graphDevice, pitch, width, height, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipFree(graphDevice));

}
 
void cudaBlockedFW(int nvertex, int *graph)
{
    HANDLE_ERROR(hipSetDevice(0));
    int *graphDevice;
    size_t height = nvertex;
    size_t width = height*sizeof(int);
    size_t pitch;

    HANDLE_ERROR(hipMallocPitch(&graphDevice, &pitch, width, height));

    HANDLE_ERROR(hipMemcpy2D(graphDevice, pitch, graph, width, width, height, hipMemcpyHostToDevice));

    dim3 gridPhase1(1 ,1, 1);
    dim3 gridPhase2((nvertex - 1) / BLOCK_SIZE + 1, 2 , 1);
    dim3 gridPhase3((nvertex - 1) / BLOCK_SIZE + 1, (nvertex - 1) / BLOCK_SIZE + 1 , 1);
    dim3 dimBlockSize(BLOCK_SIZE, BLOCK_SIZE, 1);

    int numBlock = (nvertex - 1) / BLOCK_SIZE + 1;

    for(int blockID = 0; blockID < numBlock; ++blockID) {
        // Start dependent phase
        _blocked_fw_dependent_ph<<<gridPhase1, dimBlockSize>>>(blockID, pitch / sizeof(int), nvertex, graphDevice);
        HANDLE_ERROR(hipPeekAtLastError());

        // Start partially dependent phase
        _blocked_fw_partial_dependent_ph<<<gridPhase2, dimBlockSize>>>(blockID, pitch / sizeof(int), nvertex, graphDevice);
        HANDLE_ERROR(hipPeekAtLastError());

        // Start independent phase
        _blocked_fw_independent_ph<<<gridPhase3, dimBlockSize>>>(blockID, pitch / sizeof(int), nvertex, graphDevice);
        HANDLE_ERROR(hipPeekAtLastError());
    }

    // Check for any errors launching the kernel
    HANDLE_ERROR(hipGetLastError());
    HANDLE_ERROR(hipDeviceSynchronize());

    HANDLE_ERROR(hipMemcpy2D(graph, width, graphDevice, pitch, width, height, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipFree(graphDevice));
}