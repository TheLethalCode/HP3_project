#include <hip/hip_runtime.h>


#define coarsen_factor 4
#define stride 32

/*

    Thread Level Coarsening 
    With Coarsening factor of 4 
    And Stride Length of 32
    
    Corasening Factor will reamin fixed in this file 
    Stride Length macro above can be changed for testing

*/


__global__ void SSSP_kernel1(int *V, int *E, int *W, bool *M, int *C, int *U, int n) {
    
    int tid0 = (threadIdx.x/stride)*stride*coarsen_factor  + threadIdx.x%stride + (blockIdx.x * coarsen_factor) * blockDim.x;
    
    int tid1 = tid0 + stride;
    int tid2 = tid1 + stride;
    int tid3 = tid2 + stride;
    
    if (tid0 < n && M[tid0]) {
        M[tid0] = false;
        int pos = V[tid0], size = E[pos];
        for (int i = pos + 1; i < pos + size + 1; i++) {
            int nid = E[i];
            atomicMin(&U[nid], C[tid0] + W[i]);
        }
    }

    if (tid1 < n && M[tid1]) {
        M[tid1] = false;
        int pos = V[tid1], size = E[pos];
        for (int i = pos + 1; i < pos + size + 1; i++) {
            int nid = E[i];
            atomicMin(&U[nid], C[tid1] + W[i]);
        }
    }

    if (tid2 < n && M[tid2]) {
        M[tid2] = false;
        int pos = V[tid2], size = E[pos];
        for (int i = pos + 1; i < pos + size + 1; i++) {
            int nid = E[i];
            atomicMin(&U[nid], C[tid2] + W[i]);
        }
    }

    if (tid3 < n && M[tid3]) {
        M[tid3] = false;
        int pos = V[tid3], size = E[pos];
        for (int i = pos + 1; i < pos + size + 1; i++) {
            int nid = E[i];
            atomicMin(&U[nid], C[tid3] + W[i]);
        }
    }
}

__global__ void SSSP_kernel2(bool *M, int *C, int *U, bool *flag, int n) {
    int tid0 = (threadIdx.x/stride)*stride*coarsen_factor  + threadIdx.x%stride + (blockIdx.x * coarsen_factor) * blockDim.x;
    int tid1 = tid0 + stride;
    int tid2 = tid1 + stride;
    int tid3 = tid2 + stride;

    if (tid0 < n) {
        if (C[tid0] > U[tid0]) {
            C[tid0] = U[tid0];
            M[tid0] = true;
            *flag = true;
        }
        U[tid0] = C[tid0];
    }

    if (tid1 < n) {
        if (C[tid1] > U[tid1]) {
            C[tid1] = U[tid1];
            M[tid1] = true;
            *flag = true;
        }
        U[tid1] = C[tid1];
    }

    if (tid2 < n) {
        if (C[tid2] > U[tid2]) {
            C[tid2] = U[tid2];
            M[tid2] = true;
            *flag = true;
        }
        U[tid2] = C[tid2];
    }

    if (tid3 < n) {
        if (C[tid3] > U[tid3]) {
            C[tid3] = U[tid3];
            M[tid3] = true;
            *flag = true;
        }
        U[tid3] = C[tid3];
    }
}