#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h> 
#include <>
#include <cstdio>

#include "graph.h"
#include "bfsCPU.h"

__global__ void cudaBfs(int N, int level, int *d_adjacencyList, int *d_edgesOffset,
               int *d_edgesSize, int *d_distance, int *d_parent, int *changed) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    int valueChange = 0;

    if (thid < N && d_distance[thid] == level) {
        int u = thid;
        for (int i = d_edgesOffset[u]; i < d_edgesOffset[u] + d_edgesSize[u]; i++) {
            int v = d_adjacencyList[i];
            if (level + 1 < d_distance[v]) {
                d_distance[v] = level + 1;
                d_parent[v] = i;
                valueChange = 1;
            }
        }
    }

    if (valueChange) {
        *changed = valueChange;
    }
}


void runCpu(int startVertex, Graph &G, std::vector<int> &distance,
    std::vector<int> &parent, std::vector<bool> &visited) {
printf("Starting sequential bfs.\n");
auto start = std::chrono::steady_clock::now();
bfsCPU(startVertex, G, distance, parent, visited);
auto end = std::chrono::steady_clock::now();
long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
printf("Elapsed time in milliseconds : %li ms.\n\n", duration);
}

void checkError(hipError_t error, std::string msg) {
    if (error != hipSuccess) {
        printf("%s: %d\n", msg.c_str(), error);
        exit(1);
    }
}

hipDevice_t cuDevice;
hipCtx_t cuContext;
hipModule_t cuModule;

int *d_adjacencyList;
int *d_edgesOffset;
int *d_edgesSize;
int *d_distance;
int *d_parent;

void initCuda(Graph &G) {
    //copy memory to device
    checkError(hipMalloc(&d_adjacencyList, G.numEdges * sizeof(int)), "cannot allocate d_adjacencyList");
    checkError(hipMalloc(&d_edgesOffset, G.numVertices * sizeof(int)), "cannot allocate d_edgesOffset");
    checkError(hipMalloc(&d_edgesSize, G.numVertices * sizeof(int)), "cannot allocate d_edgesSize");
    checkError(hipMalloc(&d_distance, G.numVertices * sizeof(int)), "cannot allocate d_distance");
    checkError(hipMalloc(&d_parent, G.numVertices * sizeof(int)), "cannot allocate d_parent");
    checkError(hipMemcpy(d_adjacencyList, G.adjacencyList.data(), G.numEdges * sizeof(int), hipMemcpyHostToDevice),
               "cannot copy to d_adjacencyList");
    checkError(hipMemcpy(d_edgesOffset, G.edgesOffset.data(), G.numVertices * sizeof(int), hipMemcpyHostToDevice),
               "cannot copy to d_edgesOffset");
    checkError(hipMemcpy(d_edgesSize, G.edgesSize.data(), G.numVertices * sizeof(int), hipMemcpyHostToDevice),
               "cannot copy to d_edgesSize");

}

void finalizeCuda() {
    // free memory
    checkError(hipFree(d_adjacencyList), "cannot free memory for d_adjacencyList");
    checkError(hipFree(d_edgesOffset), "cannot free memory for d_edgesOffset");
    checkError(hipFree(d_edgesSize), "cannot free memory for d_edgesSize");
    checkError(hipFree(d_distance), "cannot free memory for d_distance");
    checkError(hipFree(d_parent), "cannot free memory for d_parent");
}

void initializeCudaBfs(int startVertex, std::vector<int> &distance, std::vector<int> &parent, Graph &G) {
    //initialize values
    std::fill(distance.begin(), distance.end(), std::numeric_limits<int>::max());
    std::fill(parent.begin(), parent.end(), std::numeric_limits<int>::max());
    distance[startVertex] = 0;
    parent[startVertex] = 0;

    checkError(hipMemcpy(d_distance, distance.data(), G.numVertices * sizeof(int), hipMemcpyHostToDevice),
               "cannot copy to d)distance");
    checkError(hipMemcpy(d_parent, parent.data(), G.numVertices * sizeof(int), hipMemcpyHostToDevice),
               "cannot copy to d_parent");
}

void finalizeCudaBfs(std::vector<int> &distance, std::vector<int> &parent, Graph &G) {
    //copy memory from device
    checkError(hipMemcpy(distance.data(), d_distance, G.numVertices * sizeof(int), hipMemcpyDeviceToHost),
               "cannot copy d_distance to host");
    checkError(hipMemcpy(parent.data(), d_parent, G.numVertices * sizeof(int), hipMemcpyDeviceToHost), "cannot copy d_parent to host");

}

void runCudaSimpleBfs(int startVertex, Graph &G, std::vector<int> &distance,
                      std::vector<int> &parent) {
    initializeCudaBfs(startVertex, distance, parent, G);

    int *changed;
    checkError(hipMalloc((void **) &changed, sizeof(int)), "cannot allocate changed");

    //launch kernel
    printf("Starting simple parallel bfs.\n");
    auto start = std::chrono::steady_clock::now();

    *changed = 1;
    int level = 0;
    while (*changed) {
        *changed = 0;
        int threadsPerBlock = 1024;
        int blocksPerGrid = (G.numVertices)/ threadsPerBlock + 1;
        cudaBfs<<<blocksPerGrid, threadsPerBlock>>>(G.numVertices, level, d_adjacencyList, d_edgesOffset, d_edgesSize, d_distance, d_parent, changed);
        hipDeviceSynchronize();
        level++;
    }


    auto end = std::chrono::steady_clock::now();
    long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    printf("Elapsed time in milliseconds : %li ms.\n", duration);

    finalizeCudaBfs(distance, parent, G);
}


int main(int argc, char **argv){
    Graph G;
    readGraph(G, argc, argv);

    int startVertex = atoi(argv[1]);

    printf("Number of vertices %d\n", G.numVertices);
    printf("Number of edges %d\n\n", G.numEdges);
    //vectors for results
    std::vector<int> distance(G.numVertices, std::numeric_limits<int>::max());
    std::vector<int> parent(G.numVertices, std::numeric_limits<int>::max());
    std::vector<bool> visited(G.numVertices, false);

    //run CPU sequential bfs
    runCpu(startVertex, G, distance, parent, visited);

    initCuda(G);
    //run CUDA simple parallel bfs
    runCudaSimpleBfs(startVertex, G, distance, parent);

    finalizeCuda();
}
