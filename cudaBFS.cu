#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h> 
#include <>
#include <cstdio>

#include "graph.h"
#include "bfsCPU.h"
#include <iostream>

__global__ void cudaBfs(int N, int level, int *d_adjacencyList, int *d_edgesOffset,
               int *d_edgesSize, int *d_distance, int *d_parent, int *changed) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    int valueChange = 0;
    if (thid < N && d_distance[thid] == level) {
        int u = thid;
        for (int i = d_edgesOffset[u]; i < d_edgesOffset[u] + d_edgesSize[u]; i++) {
            int v = d_adjacencyList[i];
            if (level + 1 < d_distance[v]) {
                d_distance[v] = level + 1;
                d_parent[v] = i;
                valueChange = 1;
            }
        }
    }
    
    if (valueChange) {
        *changed = 1;
    }
}


void runCpu(int startVertex, Graph &G, std::vector<int> &distance,
    std::vector<int> &parent, std::vector<bool> &visited) {
printf("Starting sequential bfs.\n");
auto start = std::chrono::steady_clock::now();
bfsCPU(startVertex, G, distance, parent, visited);
auto end = std::chrono::steady_clock::now();
long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
printf("Elapsed time in milliseconds : %li ms.\n\n", duration);
}

void checkError(hipError_t error, std::string msg) {
    if (error != hipSuccess) {
        printf("%s: %d\n", msg.c_str(), error);
        exit(1);
    }
}

hipDevice_t cuDevice;
hipCtx_t cuContext;
hipModule_t cuModule;

int *d_adjacencyList;
int *d_edgesOffset;
int *d_edgesSize;
int *d_distance;
int *d_parent;

void initCuda(Graph &G) {
    //copy memory to device
    checkError(hipMalloc(&d_adjacencyList, G.numEdges * sizeof(int)), "cannot allocate d_adjacencyList");
    checkError(hipMalloc(&d_edgesOffset, G.numVertices * sizeof(int)), "cannot allocate d_edgesOffset");
    checkError(hipMalloc(&d_edgesSize, G.numVertices * sizeof(int)), "cannot allocate d_edgesSize");
    checkError(hipMalloc(&d_distance, G.numVertices * sizeof(int)), "cannot allocate d_distance");
    checkError(hipMalloc(&d_parent, G.numVertices * sizeof(int)), "cannot allocate d_parent");
    checkError(hipMemcpy(d_adjacencyList, G.adjacencyList.data(), G.numEdges * sizeof(int), hipMemcpyHostToDevice),
               "cannot copy to d_adjacencyList");
    checkError(hipMemcpy(d_edgesOffset, G.edgesOffset.data(), G.numVertices * sizeof(int), hipMemcpyHostToDevice),
               "cannot copy to d_edgesOffset");
    checkError(hipMemcpy(d_edgesSize, G.edgesSize.data(), G.numVertices * sizeof(int), hipMemcpyHostToDevice),
               "cannot copy to d_edgesSize");

}

void finalizeCuda() {
    // free memory
    checkError(hipFree(d_adjacencyList), "cannot free memory for d_adjacencyList");
    checkError(hipFree(d_edgesOffset), "cannot free memory for d_edgesOffset");
    checkError(hipFree(d_edgesSize), "cannot free memory for d_edgesSize");
    checkError(hipFree(d_distance), "cannot free memory for d_distance");
    checkError(hipFree(d_parent), "cannot free memory for d_parent");
}

void initializeCudaBfs(int startVertex, std::vector<int> &distance, std::vector<int> &parent, Graph &G) {
    //initialize values
    std::fill(distance.begin(), distance.end(), std::numeric_limits<int>::max());
    std::fill(parent.begin(), parent.end(), std::numeric_limits<int>::max());
    distance[startVertex] = 0;
    parent[startVertex] = 0;

    checkError(hipMemcpy(d_distance, distance.data(), G.numVertices * sizeof(int), hipMemcpyHostToDevice),
               "cannot copy to d_distance");
    checkError(hipMemcpy(d_parent, parent.data(), G.numVertices * sizeof(int), hipMemcpyHostToDevice),
               "cannot copy to d_parent");
}

void finalizeCudaBfs(std::vector<int> &distance, std::vector<int> &parent, Graph &G) {
    //copy memory from device
    checkError(hipMemcpy(distance.data(), d_distance, G.numVertices * sizeof(int), hipMemcpyDeviceToHost),
               "cannot copy d_distance to host");
    checkError(hipMemcpy(parent.data(), d_parent, G.numVertices * sizeof(int), hipMemcpyDeviceToHost), "cannot copy d_parent to host");

}

void print_array(int *arr, int N){
    std::cout << "arr :: \n";
    for (int i = 0 ; i < N; i++) std::cout << arr[i] << " ";
    std::cout << "\n";
}

void runCudaSimpleBfs(int startVertex, Graph &G, std::vector<int> &distance,
                      std::vector<int> &parent) {
    initializeCudaBfs(startVertex, distance, parent, G);

    int *changed;
    checkError(hipMalloc((void **) &changed, sizeof(int)), "cannot allocate changed");
    int zero = 0;
    int *h_changed = &zero;
    checkError(hipMemcpy(changed, h_changed, sizeof(int), hipMemcpyHostToDevice),
               "cannot copy to changed");
    //launch kernel
    printf("Starting simple parallel bfs.\n");
    auto start = std::chrono::steady_clock::now();
    
    *h_changed = 1;
    int level = 0;
    while (*h_changed) {
        *h_changed = 0;
        checkError(hipMemcpy(changed, h_changed, sizeof(int), hipMemcpyHostToDevice),
            "cannot copy to changed");
        int threadsPerBlock = (G.numVertices < 1024) ? G.numVertices : 1024;
        int blocksPerGrid = (G.numVertices + threadsPerBlock -1 )/ threadsPerBlock;
        // std::cout << threadsPerBlock << " # " << blocksPerGrid << "\n";
        cudaBfs<<<blocksPerGrid, threadsPerBlock>>>(G.numVertices, level, d_adjacencyList, d_edgesOffset, d_edgesSize, d_distance, d_parent, changed);
        // hipDeviceSynchronize();
        hipDeviceSynchronize();
        hipMemcpy(h_changed, changed, sizeof(int), hipMemcpyDeviceToHost);

        level++;
    }


    auto end = std::chrono::steady_clock::now();
    long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    printf("Elapsed time in milliseconds : %li ms.\n", duration);
    
    // int DSIZE = G.numVertices*sizeof(int);
    // int *h_data = (int *)malloc(DSIZE);
    // hipMemcpy(h_data, d_distance, DSIZE, hipMemcpyDeviceToHost);

    // print_array(h_data, G.numVertices);

    finalizeCudaBfs(distance, parent, G);
}


int main(int argc, char **argv){
    Graph G;
    readGraph(G, argc, argv);

    int startVertex = atoi(argv[1]);

    printf("Number of vertices %d\n", G.numVertices);
    printf("Number of edges %d\n\n", G.numEdges);
    //vectors for results
    std::vector<int> distance(G.numVertices, std::numeric_limits<int>::max());
    std::vector<int> parent(G.numVertices, std::numeric_limits<int>::max());
    std::vector<bool> visited(G.numVertices, false);

    //run CPU sequential bfs
    runCpu(startVertex, G, distance, parent, visited);

    initCuda(G);
    //run CUDA simple parallel bfs
    runCudaSimpleBfs(startVertex, G, distance, parent);

    finalizeCuda();
}
